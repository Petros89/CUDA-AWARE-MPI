#include "hip/hip_runtime.h"
﻿/// this program solves the 3D heat equation on a 3D structured, cartesian grid using MPI.


/**
 * The equation we want to solve can be expressed in the following way:
 
 *
 * T_t = Dx * T_xx + Dy * T_yy + Dz * T_zz,
 *
 *
 * We use a second order accurate central scheme for the space derivatives, i.e. we have (in 1D):


 * d^2 T(x) / dx^2 = T_xx ~= (T[i+1] - 2*T[i] + T[i-1]) / (dx^2)
 *
 * which we can apply in each coordinate direction equivalently. dx is the spacing between to adjacent cells, i.e. the
 * distance from one cell to its neighbors. It can be different for the y and z direction, however, within the same
 * direction it is always constant. For the time derivative, we use a first order Euler time integration scheme like so:
 *
 * dT(x) / dt = T_t ~= (T[n+1] - T[n]) / dt
 *
 * Here, n is the timestep from the previous solution and n+1 is the timestep for the next solution. In this way we can
 * integrate our solution in time. Combining the two above approximations, we could write (dor a 1D equation)
 * T_t = Dx * T_xx =>
 * (T[n+1] - T[n]) / dt = Dx * (T[i+1] - 2*T[i] + T[i-1]) / (dx^2)
 *
 * We can solve this for T[n+1] to yield:
 * T[n+1] = T[n] + (dt * Dx / (dx^2)) * (T[i+1] - 2*T[i] + T[i-1])
 *
 * We have the information of the right hand side available, thus we can calculate T[n+1] for each i.
 * For i=0 or i=iend we need to specify boundary conditions and for all T[n] we need to specify initial conditions.
 * With those information available, we can loop over time and calculate an updated solution until the solution between
 * two consequtive time steps does not change more than a user-defined convergence threshold.
 *
 * For more information on the heat equation, you may check the following link:
 * https://www.uni-muenster.de/imperia/md/content/physik_tp/lectures/ws2016-2017/num_methods_i/heat.pdf
 */

#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <algorithm>
#include <vector>
#include <array>
#include <fstream>
#include <limits>
#include <cmath>
#include <chrono>
#include <cassert>
#include "mpi.h"
#include<string>
#include <hip/hip_runtime.h>
#include <malloc.h>

#define DIM_THREAD_BLOCK_X 2
#define DIM_THREAD_BLOCK_Y 1
using namespace std;


//method to build 3D pointer
double*** CreateGrid(int m,int n,int t)
{
    int i = 0;
    int k = 0;
    double*** result = NULL; 
    if((m > 0) && (n > 0) && (t > 0))
    {
        double** pp = NULL;
        double* p = NULL;
        result = (double***)malloc(m * sizeof(double**));     
        pp = (double**)malloc(m * n * sizeof(double*));      
        p = (double*)malloc(m * n * t * sizeof(double));     
        if((result != NULL) && (pp != NULL) && (p != NULL))
        {
            for(i = 0;i < m;i++)
            {
                result[i] = pp + i * n; 
                for (k = 0;k < n;k++)
                {
                    result[i][k] = p + k * t; 
                }
                p = p + n*t;
            }
        }
        else
        {
            free(result);
            free(pp);
            free(p);
            result = NULL;
            pp = NULL;
            p = NULL;
        }
    }
    return result;
}
void FreeGrid(double*** p)
{
    if(*p != NULL)
    {
        if(**p != NULL)
        {
            free(**p);
            **p = NULL;
        }
        free(*p);
        *p = NULL;
    }
    free(p);
    p = NULL;
}

/*********************************************************************************************
                                                   GPU kernel methods
**********************************************************************************************/


__global__  void computeT(double*** TBegin, double ***TEnd, double ***Tres_gpu,int numX, int numY, int numZ, double Dx, double Dy, double Dz) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i>1&&i<(numX-2)){

	//Distribute data by X direction
	if(i<(numX)){
   
        	for (unsigned j = 1; j < numY - 1; ++j){
            		for (unsigned k = 1; i < numZ - 1; ++k) {
                		TEnd[i][j][k] = TBegin[i][j][k] +
                   		 Dx * (TBegin[i+1][j][k] - 2.0 * TBegin[i][j][k] + TBegin[i-1][j][k] )+
                   		 Dy * (TBegin[i][j+1][k] - 2.0 * TBegin[i][j][k] + TBegin[i][j-1][k]) +
                   		 Dz * (TBegin[i][j][k+1] - 2.0 * TBegin[i][j][k] + TBegin[i][j][k-1]);
				Tres_gpu[i][j][k]=TEnd[i][j][k];


            		}

		}

	}
	}

}






// based on compiler flag, use either floats or doubles for floating point operations


using floatT = double;

#define MPI_FLOAT_T MPI_DOUBLE


/// enum used to index over the respective coordinate direction
enum COORDINATE { X = 0, Y, Z };

/// enum used to access the respective direction on each local processor

/**
 *  0: LEFT
 *  1: RIGHT
 *  2: BOTTOM
 *  3: TOP
 *  4: BACK
 *  5: FRONT
 */
enum DIRECTION { LEFT = 0, RIGHT, BOTTOM, TOP, BACK, FRONT };

/// the number of physical dimensions, here 3 as we have a 3D domain

#define NUMBER_OF_DIMENSIONS 3

int main(int argc, char** argv)
{
    /// if USE_MPI is defined (see makefile), execute the following code


  /// default ranks and size (number of processors), will be rearranged by cartesian topology
 
    int rankDefaultMPICOMM, sizeDefaultMPICOMM;

    /// status and requests for non-blocking communications, i.e. MPI_IAllreduce(...) and MPI_IRecv(...)
   

    MPI_Status  status[NUMBER_OF_DIMENSIONS * 2];
    MPI_Status  postStatus[NUMBER_OF_DIMENSIONS];
    MPI_Request request[NUMBER_OF_DIMENSIONS * 2];
    MPI_Request reduceRequest;

    /// buffers into which we write data that we want to send and receive using MPI
  
    /**
     * sendbuffer will be received into receivebuffer\
    
     */
    std::array<std::vector<floatT>, NUMBER_OF_DIMENSIONS * 2> sendBuffer;
    std::array<std::vector<floatT>, NUMBER_OF_DIMENSIONS * 2> receiveBuffer;

    /// initialise MPI and get default ranks and size
    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &rankDefaultMPICOMM);
    MPI_Comm_size(MPI_COMM_WORLD, &sizeDefaultMPICOMM);

    /// new MPI communicator for cartesian topologies
   
    MPI_Comm MPI_COMM_CART;

    /// new rank and size for cartesian topology
    int       rank, size;

    /// tag used later during MPI_Send(...)
   
    int       tagSend[NUMBER_OF_DIMENSIONS * 2];

    /// tag used later during MPI_IRecv(...)
    int       tagReceive[NUMBER_OF_DIMENSIONS * 2];

    /// the dimensions are equivalent to how we want our domain to be partitioned.


    int       dimension3D[NUMBER_OF_DIMENSIONS] = { 0, 0, 0 };

    /// the coordinate in the current cartesian topology for the sub processor
    

    int       coordinates3D[NUMBER_OF_DIMENSIONS];

    /// flags to indicate if we have period boundary conditions
    
    const int periods3D[NUMBER_OF_DIMENSIONS] = { false, false, false };

    /// neighbors hold the rank of the neighboring processors and are accessed with the DIRECTION enum
   

    int       neighbors[NUMBER_OF_DIMENSIONS * 2];

    /// MPI tries to find the best possible partition of our domain and stores that in dimension3D
 

    MPI_Dims_create(sizeDefaultMPICOMM, NUMBER_OF_DIMENSIONS, dimension3D);

    /// based on the partition, we create a new cartesian topology which simplifies communication
    
    MPI_Cart_create(MPI_COMM_WORLD, NUMBER_OF_DIMENSIONS, dimension3D, periods3D, true, &MPI_COMM_CART);

    /// These calls will find the direct neighbors for each processors and return MPI_PROC_NULL if no neighbor is found.
    
    MPI_Cart_shift(MPI_COMM_CART, COORDINATE::X, 1, &neighbors[DIRECTION::LEFT], &neighbors[DIRECTION::RIGHT]);
    MPI_Cart_shift(MPI_COMM_CART, COORDINATE::Y, 1, &neighbors[DIRECTION::BOTTOM], &neighbors[DIRECTION::TOP]);
    MPI_Cart_shift(MPI_COMM_CART, COORDINATE::Z, 1, &neighbors[DIRECTION::BACK], &neighbors[DIRECTION::FRONT]);

    /// get the new rank and size for the cartesian topology
   

    MPI_Comm_rank(MPI_COMM_CART, &rank);
    MPI_Comm_size(MPI_COMM_CART, &size);

    /// get the coordinates inside our cartesian topology
 
    MPI_Cart_coords(MPI_COMM_CART, rank, NUMBER_OF_DIMENSIONS, coordinates3D);

    /// if USE_SEQUENTIAL is defined (see makefile), execute the following code
     



    /// check that we have the right number of input arguments
    
    /**
     * this is the order in which we need to pass in the command line argument:


     *
     * argv[0]: name of compiled program
     * argv[1]: number of cells in the x direction
     * argv[2]: number of cells in the y direction
     * argv[3]: number of cells in the z direction
     * argv[4]: maximum number of iterations to be used by time loop    
     * argv[5]: convergence criterion to be used to check if a solution has converged 
     */
    if (rank == 0) {
        if (argc != 6) {
            std::cout << "Incorrect number of command line arguments specified, use the following syntax:\n" << std::endl;
            std::cout << "bin/HeatEquation3D NUM_CELLS_X NUM_CELLS_Y NUM_CELLS_Z ITER_MAX EPS" << std::endl;
            std::cout << "\nor, using MPI, use the following syntax:\n" << std::endl;
            std::cout << "mpirun -n NUM_PROCS bin/HeatEquation3D NUM_CELLS_X NUM_CELLS_Y NUM_CELLS_Z ITER_MAX EPS" << std::endl;
            std::cout << "\nSee source code for additional informations!" << std::endl;
            std::abort();
        }
        else {
            std::cout << "Runnung HeatEquation3D with the following arguments: " << std::endl;
            std::cout << "executable:               " << argv[0] << std::endl;
            std::cout << "number of cells in x:     " << std::stoi(argv[1]) << std::endl;
            std::cout << "number of cells in y:     " << std::stoi(argv[2]) << std::endl;
            std::cout << "number of cells in z:     " << std::stoi(argv[3]) << std::endl;
            std::cout << "max number of iterations: " << std::stoi(argv[4]) << std::endl;


            std::cout << "convergence threshold:    " << std::stod(argv[5]) << "\n" << std::endl;

        }
    }

    /// maximum number of iterations to perform in time loop
   
    const unsigned iterMax = std::stoi(argv[4]);

    /// convergence criterion, which, once met, will terminate the calculation
    


    const floatT eps = std::stod(argv[5]);


    /// both variables are used to calculate the convergence and normalise the result.
         /**
     * We have two normalisation factors as we have to perform a reduction first (if we use MPI) to have a globally
     * available normalisation factor

          */
    floatT globalNorm = 1.0;
    floatT norm = 1.0;

    /// the break conditions used for checking of convergence has been achieved and the simulation should be stopped.
       int breakCondition = false;
    int globalBreakCondition = false;

    /// number of points (in total, not per processor) in x, y and z.
       unsigned numCells[NUMBER_OF_DIMENSIONS];
    numCells[COORDINATE::X] = std::stoi(argv[1]);
    numCells[COORDINATE::Y] = std::stoi(argv[2]);
    numCells[COORDINATE::Z] = std::stoi(argv[3]);

    /// length of the domain in x, y and z.
        floatT domainLength[NUMBER_OF_DIMENSIONS];
    domainLength[COORDINATE::X] = 1.0;
    domainLength[COORDINATE::Y] = 1.0;
    domainLength[COORDINATE::Z] = 1.0;

    /// thermal conductivity parameter. 。

    const floatT alpha = 1.0;

    /// The courant fridrich levy number                    
    const floatT CFL = 0.4;

    /// the distance between cells in the x, y and z direction.
    
    floatT spacing[NUMBER_OF_DIMENSIONS];
    spacing[COORDINATE::X] = domainLength[COORDINATE::X] / static_cast<floatT>(numCells[COORDINATE::X] - 1.0);
    spacing[COORDINATE::Y] = domainLength[COORDINATE::Y] / static_cast<floatT>(numCells[COORDINATE::Y] - 1.0);
    spacing[COORDINATE::Z] = domainLength[COORDINATE::Z] / static_cast<floatT>(numCells[COORDINATE::Z] - 1.0);

    /// the timestep to be used in the time integration.
   
    const floatT dt = CFL * 1.0 / (NUMBER_OF_DIMENSIONS * 2) *
        std::pow(std::min({ spacing[COORDINATE::X], spacing[COORDINATE::Y], spacing[COORDINATE::Z] }), 2.0) / alpha;

    /// thermal diffusivity strength in the x, y and z direction.
   

    const floatT Dx = dt * alpha / (std::pow(spacing[COORDINATE::X], 2.0));
    const floatT Dy = dt * alpha / (std::pow(spacing[COORDINATE::Y], 2.0));
    const floatT Dz = dt * alpha / (std::pow(spacing[COORDINATE::Z], 2.0));

    /// numer of iterations taken to converge solution. will be set once simulation has converged.
       unsigned finalNumIterations = 0;


    /// assure that the partition given to use by MPI can be used to partition our domain in each direction
  
    assert((numCells[COORDINATE::X] - 1) % dimension3D[COORDINATE::X] == 0 &&
        "Can not partition data for given number of processors in x!");
    assert((numCells[COORDINATE::Y] - 1) % dimension3D[COORDINATE::Y] == 0 &&
        "Can not partition data for given number of processors in y!");
    assert((numCells[COORDINATE::Z] - 1) % dimension3D[COORDINATE::Z] == 0 &&
        "Can not partition data for given number of processors in z!");

    /// chunck contains the number of cells in the x, y and z direction for each sub domain.
   

    const unsigned chunck[NUMBER_OF_DIMENSIONS] = {
      ((numCells[COORDINATE::X] - 1) / dimension3D[COORDINATE::X]) + 1,
      ((numCells[COORDINATE::Y] - 1) / dimension3D[COORDINATE::Y]) + 1,
      ((numCells[COORDINATE::Z] - 1) / dimension3D[COORDINATE::Z]) + 1
    };


    /// Create a solution vector

    std::vector<std::vector<std::vector<floatT>>> T, T0;

    /// resize both T and T0 for each sub-domain
      T.resize(chunck[COORDINATE::X]);
    T0.resize(chunck[COORDINATE::X]);
    for (unsigned i = 0; i < chunck[COORDINATE::X]; ++i) {
        T[i].resize(chunck[COORDINATE::Y]);
        T0[i].resize(chunck[COORDINATE::Y]);
        for (unsigned j = 0; j < chunck[COORDINATE::Y]; ++j) {
            T[i][j].resize(chunck[COORDINATE::Z]);
            T0[i][j].resize(chunck[COORDINATE::Z]);
        }
    }

    /// initialise each solution vector on each sub-domain with zero everywhere
       for (unsigned i = 0; i < chunck[COORDINATE::X]; ++i)
        for (unsigned j = 0; j < chunck[COORDINATE::Y]; ++j)
            for (unsigned k = 0; k < chunck[COORDINATE::Z]; ++k)
                T[i][j][k] = 0.0;

    /// apply boundary conditions on the top of the domain
    
    if (neighbors[DIRECTION::TOP] == MPI_PROC_NULL)

        for (unsigned i = 0; i < chunck[COORDINATE::X]; ++i)
            for (unsigned k = 0; k < chunck[COORDINATE::Z]; ++k)
                T[i][chunck[COORDINATE::Y] - 1][k] = 1.0;

    /// apply boundary conditions on the left-side of the domain
     

    if (neighbors[DIRECTION::LEFT] == MPI_PROC_NULL)

        for (unsigned j = 0; j < chunck[COORDINATE::Y]; ++j)
            for (unsigned k = 0; k < chunck[COORDINATE::Z]; ++k)
                T[0][j][k] = (coordinates3D[COORDINATE::Y] * (chunck[COORDINATE::Y] - 1) + j) * spacing[COORDINATE::Y];

    /// apply boundary conditions on the right-side of the domain
      
    if (neighbors[DIRECTION::RIGHT] == MPI_PROC_NULL)

        for (unsigned j = 0; j < chunck[COORDINATE::Y]; ++j)
            for (unsigned k = 0; k < chunck[COORDINATE::Z]; ++k)
                T[chunck[COORDINATE::X] - 1][j][k] = (coordinates3D[COORDINATE::Y] * (chunck[COORDINATE::Y] - 1) + j) * spacing[COORDINATE::Y];

    /// apply boundary conditions on the back-side of the domain
    
    if (neighbors[DIRECTION::BACK] == MPI_PROC_NULL)

        for (unsigned i = 0; i < chunck[COORDINATE::X]; ++i)
            for (unsigned j = 0; j < chunck[COORDINATE::Y]; ++j)
                T[i][j][0] = (coordinates3D[COORDINATE::Y] * (chunck[COORDINATE::Y] - 1) + j) * spacing[COORDINATE::Y];

    /// apply boundary conditions on the front-side of the domain
     
    if (neighbors[DIRECTION::FRONT] == MPI_PROC_NULL)

        for (unsigned i = 0; i < chunck[COORDINATE::X]; ++i)
            for (unsigned j = 0; j < chunck[COORDINATE::Y]; ++j)
                T[i][j][chunck[COORDINATE::Z] - 1] = (coordinates3D[COORDINATE::Y] * (chunck[COORDINATE::Y] - 1) + j) * spacing[COORDINATE::Y];

    /// if we use MPI, make sure that our send and recieve buffers are correctly allocated
    

  /// allocate storage for left-side send- and recievebuffer
  
    if (neighbors[DIRECTION::LEFT] != MPI_PROC_NULL) {
        sendBuffer[DIRECTION::LEFT].resize((chunck[COORDINATE::Y] - 1) * (chunck[COORDINATE::Z] - 1));
        receiveBuffer[DIRECTION::LEFT].resize((chunck[COORDINATE::Y] - 1) * (chunck[COORDINATE::Z] - 1));
    }
    else {
        sendBuffer[DIRECTION::LEFT].resize(1);
        receiveBuffer[DIRECTION::LEFT].resize(1);
    }

    /// allocate storage for right-side send- and recievebuffer
   
    if (neighbors[DIRECTION::RIGHT] != MPI_PROC_NULL) {
        sendBuffer[DIRECTION::RIGHT].resize((chunck[COORDINATE::Y] - 1) * (chunck[COORDINATE::Z] - 1));
        receiveBuffer[DIRECTION::RIGHT].resize((chunck[COORDINATE::Y] - 1) * (chunck[COORDINATE::Z] - 1));
    }
    else {
        sendBuffer[DIRECTION::RIGHT].resize(1);
        receiveBuffer[DIRECTION::RIGHT].resize(1);
    }

    /// allocate storage for bottom-side send- and recievebuffer
   
    if (neighbors[DIRECTION::BOTTOM] != MPI_PROC_NULL) {
        sendBuffer[DIRECTION::BOTTOM].resize((chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Z] - 1));
        receiveBuffer[DIRECTION::BOTTOM].resize((chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Z] - 1));
    }
    else {
        sendBuffer[DIRECTION::BOTTOM].resize(1);
        receiveBuffer[DIRECTION::BOTTOM].resize(1);
    }

    /// allocate storage for top-side send- and recievebuffer
    
    if (neighbors[DIRECTION::TOP] != MPI_PROC_NULL) {
        sendBuffer[DIRECTION::TOP].resize((chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Z] - 1));
        receiveBuffer[DIRECTION::TOP].resize((chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Z] - 1));
    }
    else {
        sendBuffer[DIRECTION::TOP].resize(1);
        receiveBuffer[DIRECTION::TOP].resize(1);

    }

    /// allocate storage for back-side send- and recievebuffer
    
    if (neighbors[DIRECTION::BACK] != MPI_PROC_NULL) {
        sendBuffer[DIRECTION::BACK].resize((chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Y] - 1));
        receiveBuffer[DIRECTION::BACK].resize((chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Y] - 1));
    }
    else {
        sendBuffer[DIRECTION::BACK].resize(1);
        receiveBuffer[DIRECTION::BACK].resize(1);
    }

    /// allocate storage for front-side send- and recievebuffer
   
    if (neighbors[DIRECTION::FRONT] != MPI_PROC_NULL) {
        sendBuffer[DIRECTION::FRONT].resize((chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Y] - 1));
        receiveBuffer[DIRECTION::FRONT].resize((chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Y] - 1));
    }
    else {
        sendBuffer[DIRECTION::FRONT].resize(1);
        receiveBuffer[DIRECTION::FRONT].resize(1);
    }

    /// start timing (we don't want any setup time to be included, thus we start it just before the time loop)
   
    auto start = MPI_Wtime();


    /// main time loop
    /**
     * this is where we solve the actual partial differential equation and do the communication among processors.
          */



   



    for (unsigned time = 0; time < iterMax; ++time)
    {
        /// copy the solution from the previous timestep into T, which holds the solution of the last iteration
         
        for (unsigned i = 0; i < chunck[COORDINATE::X]; ++i)
            for (unsigned j = 0; j < chunck[COORDINATE::Y]; ++j)
                for (unsigned k = 0; k < chunck[COORDINATE::Z]; ++k)
                    T0[i][j][k] = T[i][j][k];

        // HALO communication step



  /// preparing the send buffer (the data we want to send to the left neighbor), if a neighbor exists


  /**
   * for simplicity, we write the 2D array (the face on the boundary) into a 1D array which we can easily send.
   * It is important that once we receive the it we are aware that the array containing the data is 1D now.

   */
        unsigned counter = 0;
        if (neighbors[DIRECTION::LEFT] != MPI_PROC_NULL)
            for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j)
                for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k)
                    sendBuffer[DIRECTION::LEFT][counter++] = T0[1][j][k];

        /// preparing the send buffer (the data we want to send to the right neighbor), if a neighbor exists
       
        counter = 0;
        if (neighbors[DIRECTION::RIGHT] != MPI_PROC_NULL)
            for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j)
                for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k)
                    sendBuffer[DIRECTION::RIGHT][counter++] = T0[chunck[COORDINATE::X] - 2][j][k];

        /// preparing the send buffer (the data we want to send to the bottom neighbor), if a neighbor exists
                counter = 0;
        if (neighbors[DIRECTION::BOTTOM] != MPI_PROC_NULL)
            for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
                for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k)
                    sendBuffer[DIRECTION::BOTTOM][counter++] = T0[i][1][k];

        /// preparing the send buffer (the data we want to send to the top neighbor), if a neighbor exists
      

        counter = 0;
        if (neighbors[DIRECTION::TOP] != MPI_PROC_NULL)
            for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
                for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k)
                    sendBuffer[DIRECTION::TOP][counter++] = T0[i][chunck[COORDINATE::Y] - 2][k];

        /// preparing the send buffer (the data we want to send to the back neighbor), if a neighbor exists
                counter = 0;
        if (neighbors[DIRECTION::BACK] != MPI_PROC_NULL)
            for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
                for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j)
                    sendBuffer[DIRECTION::BACK][counter++] = T0[i][j][1];

        /// preparing the send buffer (the data we want to send to the front neighbor), if a neighbor exists
       
        counter = 0;
        if (neighbors[DIRECTION::FRONT] != MPI_PROC_NULL)
            for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
                for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j)
                    sendBuffer[DIRECTION::FRONT][counter++] = T0[i][j][chunck[COORDINATE::Z] - 2];


       

        /// prepare the tags we need to append to the send message for each send (in each direction) and receive
        

        for (unsigned index = 0; index < NUMBER_OF_DIMENSIONS * 2; ++index) {
            tagSend[index] = 100 + neighbors[index];
            tagReceive[index] = 100 + rank;
        }

        /// send the prepared send buffer to the neighbors using non-blocking MPI_Isend(...)
                MPI_Isend(&sendBuffer[DIRECTION::LEFT][0], (chunck[COORDINATE::Y] - 1) * (chunck[COORDINATE::Z] - 1),
            MPI_FLOAT_T, neighbors[DIRECTION::LEFT], tagSend[DIRECTION::LEFT], MPI_COMM_CART,
            &request[DIRECTION::LEFT]);

        MPI_Isend(&sendBuffer[DIRECTION::RIGHT][0], (chunck[COORDINATE::Y] - 1) * (chunck[COORDINATE::Z] - 1),
            MPI_FLOAT_T, neighbors[DIRECTION::RIGHT], tagSend[DIRECTION::RIGHT], MPI_COMM_CART,
            &request[DIRECTION::RIGHT]);

        MPI_Isend(&sendBuffer[DIRECTION::BOTTOM][0], (chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Z] - 1),
            MPI_FLOAT_T, neighbors[DIRECTION::BOTTOM], tagSend[DIRECTION::BOTTOM], MPI_COMM_CART,
            &request[DIRECTION::BOTTOM]);

        MPI_Isend(&sendBuffer[DIRECTION::TOP][0], (chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Z] - 1),
            MPI_FLOAT_T, neighbors[DIRECTION::TOP], tagSend[DIRECTION::TOP], MPI_COMM_CART,
            &request[DIRECTION::TOP]);

        MPI_Isend(&sendBuffer[DIRECTION::BACK][0], (chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Y] - 1),
            MPI_FLOAT_T, neighbors[DIRECTION::BACK], tagSend[DIRECTION::BACK], MPI_COMM_CART,
            &request[DIRECTION::BACK]);

        MPI_Isend(&sendBuffer[DIRECTION::FRONT][0], (chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Y] - 1),
            MPI_FLOAT_T, neighbors[DIRECTION::FRONT], tagSend[DIRECTION::FRONT], MPI_COMM_CART,
            &request[DIRECTION::FRONT]);


        /*****************************************************************************************************************
                                                          GPU BEGIN
   ****************************************************************************************************************/
        // compute internal domain (no halos required)
        
	//Assign GPU to each process
	int deviceCount;

  	hipGetDeviceCount(&deviceCount);
  	int device_id = rank%deviceCount;
  	hipSetDevice(device_id); 
	
        int numX = chunck[COORDINATE::X];
        int numY = chunck[COORDINATE::Y];
        int numZ = chunck[COORDINATE::X];
	int num = numX*numY*numZ;
        double ***TBegin = CreateGrid(numX, numY, numZ);
	double ***TEnd = CreateGrid(numX, numY, numZ);
	double ***Tres = CreateGrid(numX, numY, numZ);
	double ***Tres_gpu = CreateGrid(numX, numY, numZ);
	double ***Thost = CreateGrid(numX, numY, numZ);
	double ***Thost0 = CreateGrid(numX, numY, numZ);
       
	for (unsigned i = 0; i < chunck[COORDINATE::X]; ++i){
            for (unsigned j = 0; j < chunck[COORDINATE::Y]; ++j){
                for (unsigned k = 0; k < chunck[COORDINATE::Z]; ++k){
                    *(*(*(Thost0 + i) + j) + k)=T0[i][j][k];
			*(*(*(Thost + i) + j) + k)=T[i][j][k];
	//printf("Thost0: %f", Thost0[i][j][k]);
	//printf("\n");


		}
	    }
	}
	
	//malloc memory 
        hipMalloc((void**)&TBegin, sizeof(double) * num);
        hipMalloc((void**)&TEnd, sizeof(double) * num);
	hipMalloc((void**)&Tres_gpu, sizeof(double) * num);


        hipMemcpy(TBegin, Thost0, sizeof(double) * num, hipMemcpyHostToDevice);
        hipMemcpy(TEnd, Thost, sizeof(double) * num, hipMemcpyHostToDevice);
	
	//set grid and block size
        dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
        dim3 grid((size_t)ceil((double)(numX-2)/ ((double)DIM_THREAD_BLOCK_X)), 1);
	
	//gpu compute kernel
        computeT <<<grid,block>>> (TBegin, TEnd, Tres_gpu,numX, numY, numZ, Dx, Dy,Dz);
        
	//pass data from gpu to cpu
        hipMemcpy(Tres, Tres_gpu, sizeof(double) *num, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

        for (unsigned i = 0; i < chunck[COORDINATE::X]; ++i){
            for (unsigned j = 0; j < chunck[COORDINATE::Y]; ++j){
                for (unsigned k = 0; k < chunck[COORDINATE::Z]; ++k){
                    T[i][j][k]=Tres[i][j][k];
			
		}
	    }
	}
	FreeGrid(TBegin);
	FreeGrid(TEnd);
	FreeGrid(Tres);
	FreeGrid(Tres_gpu);
	FreeGrid(Thost);
	FreeGrid(Thost0);
	hipFree(TBegin);
	hipFree(TEnd);
	hipFree(Tres_gpu);

        /// now work on the halo cells
       

  /// receive the halo information from each neighbor, if exists.


        MPI_Recv(&receiveBuffer[DIRECTION::LEFT][0], (chunck[COORDINATE::Y] - 1) * (chunck[COORDINATE::Z] - 1),
            MPI_FLOAT_T, neighbors[DIRECTION::LEFT], tagReceive[DIRECTION::LEFT], MPI_COMM_CART,
            &status[DIRECTION::LEFT]);

        MPI_Recv(&receiveBuffer[DIRECTION::RIGHT][0], (chunck[COORDINATE::Y] - 1) * (chunck[COORDINATE::Z] - 1),
            MPI_FLOAT_T, neighbors[DIRECTION::RIGHT], tagReceive[DIRECTION::RIGHT], MPI_COMM_CART,
            &status[DIRECTION::RIGHT]);

        MPI_Recv(&receiveBuffer[DIRECTION::BOTTOM][0], (chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Z] - 1),
            MPI_FLOAT_T, neighbors[DIRECTION::BOTTOM], tagReceive[DIRECTION::BOTTOM], MPI_COMM_CART,
            &status[DIRECTION::BOTTOM]);

        MPI_Recv(&receiveBuffer[DIRECTION::TOP][0], (chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Z] - 1),
            MPI_FLOAT_T, neighbors[DIRECTION::TOP], tagReceive[DIRECTION::TOP], MPI_COMM_CART,
            &status[DIRECTION::TOP]);

        MPI_Recv(&receiveBuffer[DIRECTION::BACK][0], (chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Y] - 1),
            MPI_FLOAT_T, neighbors[DIRECTION::BACK], tagReceive[DIRECTION::BACK], MPI_COMM_CART,
            &status[DIRECTION::BACK]);

        MPI_Recv(&receiveBuffer[DIRECTION::FRONT][0], (chunck[COORDINATE::X] - 1) * (chunck[COORDINATE::Y] - 1),
            MPI_FLOAT_T, neighbors[DIRECTION::FRONT], tagReceive[DIRECTION::FRONT], MPI_COMM_CART,
            &status[DIRECTION::FRONT]);

        /// make sure that all communications have been executed
        
        /**
         * even though we use a blocking receive here, since we used a non-blocking send, we have to wait for all
         * communications to have finished before continuing.
        
         */
        MPI_Waitall(NUMBER_OF_DIMENSIONS * 2, request, status);

        /// now that we have the halo cells, we update the boundaries using information from other processors
      

        if (neighbors[DIRECTION::LEFT] != MPI_PROC_NULL) {
            const auto& THalo = receiveBuffer[DIRECTION::LEFT];
            unsigned i = 0;
            unsigned counter = 0;

            for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j)
                for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k) {
                    T[i][j][k] = T0[i][j][k] +
                        Dx * (T0[i + 1][j][k] - 2.0 * T0[i][j][k] + THalo[counter++]) +
                        Dy * (T0[i][j + 1][k] - 2.0 * T0[i][j][k] + T0[i][j - 1][k]) +
                        Dz * (T0[i][j][k + 1] - 2.0 * T0[i][j][k] + T0[i][j][k - 1]);
                }
        }

        /// do the same as above, this time for the right neighbor halo data
      

        if (neighbors[DIRECTION::RIGHT] != MPI_PROC_NULL) {
            const auto& THalo = receiveBuffer[DIRECTION::RIGHT];
            unsigned i = chunck[COORDINATE::X] - 1;
            unsigned counter = 0;

            for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j)
                for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k) {
                    T[i][j][k] = T0[i][j][k] +
                        Dx * (THalo[counter++] - 2.0 * T0[i][j][k] + T0[i - 1][j][k]) +
                        Dy * (T0[i][j + 1][k] - 2.0 * T0[i][j][k] + T0[i][j - 1][k]) +
                        Dz * (T0[i][j][k + 1] - 2.0 * T0[i][j][k] + T0[i][j][k - 1]);
                }
        }

        /// do the same as above, this time for the bottom neighbor halo data
       
        if (neighbors[DIRECTION::BOTTOM] != MPI_PROC_NULL) {
            const auto& THalo = receiveBuffer[DIRECTION::BOTTOM];
            unsigned j = 0;
            unsigned counter = 0;

            for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
                for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k) {
                    T[i][j][k] = T0[i][j][k] +
                        Dx * (T0[i + 1][j][k] - 2.0 * T0[i][j][k] + T0[i - 1][j][k]) +
                        Dy * (T0[i][j + 1][k] - 2.0 * T0[i][j][k] + THalo[counter++]) +
                        Dz * (T0[i][j][k + 1] - 2.0 * T0[i][j][k] + T0[i][j][k - 1]);
                }
        }

        /// do the same as above, this time for the top neighbor halo data
      
        if (neighbors[DIRECTION::TOP] != MPI_PROC_NULL) {
            const auto& THalo = receiveBuffer[DIRECTION::TOP];
            unsigned j = chunck[COORDINATE::Y] - 1;
            unsigned counter = 0;

            for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
                for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k) {
                    T[i][j][k] = T0[i][j][k] +
                        Dx * (T0[i + 1][j][k] - 2.0 * T0[i][j][k] + T0[i - 1][j][k]) +
                        Dy * (THalo[counter++] - 2.0 * T0[i][j][k] + T0[i][j - 1][k]) +
                        Dz * (T0[i][j][k + 1] - 2.0 * T0[i][j][k] + T0[i][j][k - 1]);
                }
        }

        /// do the same as above, this time for the back neighbor halo data
        
        if (neighbors[DIRECTION::BACK] != MPI_PROC_NULL) {
            const auto& THalo = receiveBuffer[DIRECTION::BACK];
            unsigned k = 0;
            unsigned counter = 0;

            for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
                for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j) {
                    T[i][j][k] = T0[i][j][k] +
                        Dx * (T0[i + 1][j][k] - 2.0 * T0[i][j][k] + T0[i - 1][j][k]) +
                        Dy * (T0[i][j + 1][k] - 2.0 * T0[i][j][k] + T0[i][j - 1][k]) +
                        Dz * (T0[i][j][k + 1] - 2.0 * T0[i][j][k] + THalo[counter++]);
                }
        }

        /// do the same as above, this time for the front neighbor halo data
       
        if (neighbors[DIRECTION::FRONT] != MPI_PROC_NULL) {
            const auto& THalo = receiveBuffer[DIRECTION::FRONT];
            unsigned k = chunck[COORDINATE::Z] - 1;
            unsigned counter = 0;

            for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
                for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j) {
                    T[i][j][k] = T0[i][j][k] +
                        Dx * (T0[i + 1][j][k] - 2.0 * T0[i][j][k] + T0[i - 1][j][k]) +
                        Dy * (T0[i][j + 1][k] - 2.0 * T0[i][j][k] + T0[i][j - 1][k]) +
                        Dz * (THalo[counter++] - 2.0 * T0[i][j][k] + T0[i][j][k - 1]);
                }
        }
        /************************************************************************************************************

                                                                GPU      END

       ***********************************************************************************************************/
        /// update edges of halo elements
       
        if (neighbors[DIRECTION::LEFT] != MPI_PROC_NULL) {
            if (neighbors[DIRECTION::BOTTOM] != MPI_PROC_NULL) {
                unsigned i = 0;
                unsigned j = 0;
                for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k)
                    T[i][j][k] = 2.0 * T[i + 1][j][k] - T[i + 2][j][k];
            }
            if (neighbors[DIRECTION::TOP] != MPI_PROC_NULL) {
                unsigned i = 0;
                unsigned j = chunck[COORDINATE::Y] - 1;
                for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k)
                    T[i][j][k] = 2.0 * T[i + 1][j][k] - T[i + 2][j][k];
            }
            if (neighbors[DIRECTION::BACK] != MPI_PROC_NULL) {
                unsigned i = 0;
                unsigned k = 0;
                for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j)
                    T[i][j][k] = 2.0 * T[i + 1][j][k] - T[i + 2][j][k];
            }
            if (neighbors[DIRECTION::FRONT] != MPI_PROC_NULL) {
                unsigned i = 0;
                unsigned k = chunck[COORDINATE::Z] - 1;
                for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j)
                    T[i][j][k] = 2.0 * T[i + 1][j][k] - T[i + 2][j][k];
            }
        }

        if (neighbors[DIRECTION::RIGHT] != MPI_PROC_NULL) {
            if (neighbors[DIRECTION::BOTTOM] != MPI_PROC_NULL) {
                unsigned i = chunck[COORDINATE::X] - 1;
                unsigned j = 0;
                for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k)
                    T[i][j][k] = 2.0 * T[i - 1][j][k] - T[i - 2][j][k];
            }
            if (neighbors[DIRECTION::TOP] != MPI_PROC_NULL) {
                unsigned i = chunck[COORDINATE::X] - 1;
                unsigned j = chunck[COORDINATE::Y] - 1;
                for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k)
                    T[i][j][k] = 2.0 * T[i - 1][j][k] - T[i - 2][j][k];
            }
            if (neighbors[DIRECTION::BACK] != MPI_PROC_NULL) {
                unsigned i = chunck[COORDINATE::X] - 1;
                unsigned k = 0;
                for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j)
                    T[i][j][k] = 2.0 * T[i - 1][j][k] - T[i - 2][j][k];
            }
            if (neighbors[DIRECTION::FRONT] != MPI_PROC_NULL) {
                unsigned i = chunck[COORDINATE::X] - 1;
                unsigned k = chunck[COORDINATE::Z] - 1;
                for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j)
                    T[i][j][k] = 2.0 * T[i - 1][j][k] - T[i - 2][j][k];
            }
        }

        if (neighbors[DIRECTION::BACK] != MPI_PROC_NULL) {
            if (neighbors[DIRECTION::BOTTOM] != MPI_PROC_NULL) {
                unsigned j = 0;
                unsigned k = 0;
                for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
                    T[i][j][k] = 2.0 * T[i][j][k + 1] - T[i][j][k + 2];
            }
            if (neighbors[DIRECTION::TOP] != MPI_PROC_NULL) {
                unsigned j = chunck[COORDINATE::Y] - 1;
                unsigned k = 0;
                for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
                    T[i][j][k] = 2.0 * T[i][j][k + 1] - T[i][j][k + 2];
            }
        }

        if (neighbors[DIRECTION::FRONT] != MPI_PROC_NULL) {
            if (neighbors[DIRECTION::BOTTOM] != MPI_PROC_NULL) {
                unsigned j = 0;
                unsigned k = chunck[COORDINATE::Z] - 1;
                for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
                    T[i][j][k] = 2.0 * T[i][j][k - 1] - T[i][j][k - 2];
            }
            if (neighbors[DIRECTION::TOP] != MPI_PROC_NULL) {
                unsigned j = chunck[COORDINATE::Y] - 1;
                unsigned k = chunck[COORDINATE::Z] - 1;
                for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
                    T[i][j][k] = 2.0 * T[i][j][k - 1] - T[i][j][k - 2];
            }
        }
        /// finished with halo edges extrapolation
       

        /// at last, we update the boundary points through weighted averages
      
        if ((neighbors[DIRECTION::LEFT] != MPI_PROC_NULL) && (neighbors[DIRECTION::BOTTOM] != MPI_PROC_NULL) &&
            (neighbors[DIRECTION::BACK] != MPI_PROC_NULL)) {
            unsigned i = 0;
            unsigned j = 0;
            unsigned k = 0;
            T[i][j][k] = 1.0 / 3.0 * (T[i + 1][j][k] + T[i][j + 1][k] + T[i][j][k + 1]);
        }

        if ((neighbors[DIRECTION::LEFT] != MPI_PROC_NULL) && (neighbors[DIRECTION::BOTTOM] != MPI_PROC_NULL) &&
            (neighbors[DIRECTION::FRONT] != MPI_PROC_NULL)) {
            unsigned i = 0;
            unsigned j = 0;
            unsigned k = chunck[COORDINATE::Z] - 1;
            T[i][j][k] = 1.0 / 3.0 * (T[i + 1][j][k] + T[i][j + 1][k] + T[i][j][k - 1]);
        }

        if ((neighbors[DIRECTION::LEFT] != MPI_PROC_NULL) && (neighbors[DIRECTION::TOP] != MPI_PROC_NULL) &&
            (neighbors[DIRECTION::BACK] != MPI_PROC_NULL)) {
            unsigned i = 0;
            unsigned j = chunck[COORDINATE::Y] - 1;
            unsigned k = 0;
            T[i][j][k] = 1.0 / 3.0 * (T[i + 1][j][k] + T[i][j - 1][k] + T[i][j][k + 1]);
        }

        if ((neighbors[DIRECTION::LEFT] != MPI_PROC_NULL) && (neighbors[DIRECTION::TOP] != MPI_PROC_NULL) &&
            (neighbors[DIRECTION::FRONT] != MPI_PROC_NULL)) {
            unsigned i = 0;
            unsigned j = chunck[COORDINATE::Y] - 1;
            unsigned k = chunck[COORDINATE::Z] - 1;
            T[i][j][k] = 1.0 / 3.0 * (T[i + 1][j][k] + T[i][j - 1][k] + T[i][j][k - 1]);
        }

        if ((neighbors[DIRECTION::RIGHT] != MPI_PROC_NULL) && (neighbors[DIRECTION::BOTTOM] != MPI_PROC_NULL) &&
            (neighbors[DIRECTION::BACK] != MPI_PROC_NULL)) {
            unsigned i = chunck[COORDINATE::X] - 1;
            unsigned j = 0;
            unsigned k = 0;
            T[i][j][k] = 1.0 / 3.0 * (T[i - 1][j][k] + T[i][j + 1][k] + T[i][j][k + 1]);
        }

        if ((neighbors[DIRECTION::RIGHT] != MPI_PROC_NULL) && (neighbors[DIRECTION::BOTTOM] != MPI_PROC_NULL) &&
            (neighbors[DIRECTION::FRONT] != MPI_PROC_NULL)) {
            unsigned i = chunck[COORDINATE::X] - 1;
            unsigned j = 0;
            unsigned k = chunck[COORDINATE::Z] - 1;
            T[i][j][k] = 1.0 / 3.0 * (T[i - 1][j][k] + T[i][j + 1][k] + T[i][j][k - 1]);
        }

        if ((neighbors[DIRECTION::RIGHT] != MPI_PROC_NULL) && (neighbors[DIRECTION::TOP] != MPI_PROC_NULL) &&
            (neighbors[DIRECTION::BACK] != MPI_PROC_NULL)) {
            unsigned i = chunck[COORDINATE::X] - 1;
            unsigned j = chunck[COORDINATE::Y] - 1;
            unsigned k = 0;
            T[i][j][k] = 1.0 / 3.0 * (T[i - 1][j][k] + T[i][j - 1][k] + T[i][j][k + 1]);
        }

        if ((neighbors[DIRECTION::RIGHT] != MPI_PROC_NULL) && (neighbors[DIRECTION::TOP] != MPI_PROC_NULL) &&
            (neighbors[DIRECTION::FRONT] != MPI_PROC_NULL)) {
            unsigned i = chunck[COORDINATE::X] - 1;
            unsigned j = chunck[COORDINATE::Y] - 1;
            unsigned k = chunck[COORDINATE::Z] - 1;
            T[i][j][k] = 1.0 / 3.0 * (T[i - 1][j][k] + T[i][j - 1][k] + T[i][j][k - 1]);
        }
        /// finished with halo corner points
     


/// calculate the difference between the current and previous (last time step) solution.
  

        floatT res = std::numeric_limits<floatT>::min();
        for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
            for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j)
                for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k)
                    if (std::fabs(T[i][j][k] - T0[i][j][k]) > res)
                        res = std::fabs(T[i][j][k] - T0[i][j][k]);

        /// if it is the first time step, store the residual as the normalisation factor
      


        if (time == 0)
            if (res != 0.0)
                norm = res;

        /// For MPI, we have to communicate the norm by selecting the lowest among all processors
       

        if (time == 0) {
            MPI_Iallreduce(&norm, &globalNorm, 1, MPI_FLOAT_T, MPI_MIN, MPI_COMM_CART, &reduceRequest);
            MPI_Wait(&reduceRequest, MPI_STATUS_IGNORE);
        }


        /// if we want to debug, it may be useful to see the residuals. Turned of for release builds for performance.
          
//#if defined(USE_DEBUG)
//        if (rank == 0) {
//            std::cout << "time: " << std::setw(10) << time;
//            std::cout << std::scientific << std::setw(15) << std::setprecision(5) << ", residual: ";
//            std::cout << res / norm << std::endl;
//        }
//#endif

        /// check if the current residual has dropped below our defined convergence threshold "eps"
         
        if (res / norm < eps)
            breakCondition = true;

        /// Again, for MPI we need to among all processors if we can break from the loop
       


        MPI_Iallreduce(&breakCondition, &globalBreakCondition, 1, MPI_INT, MPI_MAX, MPI_COMM_CART, &reduceRequest);
        MPI_Wait(&reduceRequest, MPI_STATUS_IGNORE);


        /// final check if we can break, the above was just preparation for this check.
         


        if (globalBreakCondition) {
            finalNumIterations = time;
            break;
        }
    }
    /// done with the time loop
   

    /// output the timing information to screen.
    

    auto end = MPI_Wtime();
    if (rank == 0) {
        std::cout << "Computational time (parallel): " << std::fixed << (end - start) << "\n" << std::endl;
        if (globalBreakCondition) {
            std::cout << "Simulation has converged in " << finalNumIterations << " iterations";
            std::cout << " with a convergence threshold of " << std::scientific << eps << std::endl;
        }
        else
            std::cout << "Simulation did not converge within " << iterMax << " iterations." << std::endl;
    }


    /// calculate the error we have made against the analytic solution
   

    double globalError = 0.0;
    double error = 0.0;
    for (unsigned k = 1; k < chunck[COORDINATE::Z] - 1; ++k)
        for (unsigned j = 1; j < chunck[COORDINATE::Y] - 1; ++j)
            for (unsigned i = 1; i < chunck[COORDINATE::X] - 1; ++i)
                error += std::sqrt(std::pow(T[i][j][k] - (coordinates3D[COORDINATE::Y] * (chunck[COORDINATE::Y] - 1) + j) * spacing[COORDINATE::Y], 2.0));
    error /= ((chunck[COORDINATE::X] - 2) * (chunck[COORDINATE::Y] - 2) * (chunck[COORDINATE::Z] - 2));
    MPI_Iallreduce(&error, &globalError, 1, MPI_FLOAT_T, MPI_SUM, MPI_COMM_CART, &reduceRequest);
    MPI_Wait(&reduceRequest, MPI_STATUS_IGNORE);
    if (rank == 0)
        std::cout << "L2-norm error: " << std::fixed << std::setprecision(4) << 100 * error << " %" << std::endl;


    /// output the solution in a format readable by a post processor, such as paraview.
     

    std::vector<floatT> receiveBufferPostProcess;
    receiveBufferPostProcess.resize(chunck[COORDINATE::X] * chunck[COORDINATE::Y] * chunck[COORDINATE::Z]);
    if (rank > 0 && size != 1)
    {
        int counter = 0;
        for (unsigned k = 0; k < chunck[COORDINATE::Z]; ++k)
            for (unsigned j = 0; j < chunck[COORDINATE::Y]; ++j)
                for (unsigned i = 0; i < chunck[COORDINATE::X]; ++i)
                    receiveBufferPostProcess[counter++] = T[i][j][k];

        MPI_Send(&receiveBufferPostProcess[0], chunck[COORDINATE::X] * chunck[COORDINATE::Y] * chunck[COORDINATE::Z], MPI_FLOAT_T, 0, 200 + rank, MPI_COMM_CART);
        MPI_Send(&coordinates3D[0], NUMBER_OF_DIMENSIONS, MPI_INT, 0, 300 + rank, MPI_COMM_CART);
    }
    if (rank == 0 && size != 1)
    {
        std::ofstream out("output/out.dat");
        out << "TITLE=\"out\"" << std::endl;
        out << "VARIABLES = \"X\", \"Y\", \"Z\", \"T\", \"rank\"" << std::endl;
        out << "ZONE T = \"" << rank << "\", I=" << chunck[COORDINATE::X] << ", J=" << chunck[COORDINATE::Y] << ", K=" << chunck[COORDINATE::Z] << ", F=POINT" << std::endl;
        for (unsigned k = 0; k < chunck[COORDINATE::Z]; ++k)
            for (unsigned j = 0; j < chunck[COORDINATE::Y]; ++j)
                for (unsigned i = 0; i < chunck[COORDINATE::X]; ++i)
                {
                    out << std::scientific << std::setprecision(5) << std::setw(15) << (coordinates3D[COORDINATE::X] * (chunck[COORDINATE::X] - 1) + i) * spacing[COORDINATE::X];
                    out << std::scientific << std::setprecision(5) << std::setw(15) << (coordinates3D[COORDINATE::Y] * (chunck[COORDINATE::Y] - 1) + j) * spacing[COORDINATE::Y];
                    out << std::scientific << std::setprecision(5) << std::setw(15) << (coordinates3D[COORDINATE::Z] * (chunck[COORDINATE::Z] - 1) + k) * spacing[COORDINATE::Z];
                    out << std::scientific << std::setprecision(5) << std::setw(15) << T[i][j][k];
                    out << std::fixed << std::setw(5) << rank << std::endl;
                }

        for (int recvRank = 1; recvRank < size; ++recvRank)
        {
            int coordinates3DFromReceivedRank[NUMBER_OF_DIMENSIONS];
            MPI_Recv(&receiveBufferPostProcess[0], chunck[COORDINATE::X] * chunck[COORDINATE::Y] * chunck[COORDINATE::Z], MPI_FLOAT_T, recvRank, 200 + recvRank, MPI_COMM_CART, &postStatus[0]);
            MPI_Recv(&coordinates3DFromReceivedRank[0], NUMBER_OF_DIMENSIONS, MPI_INT, recvRank, 300 + recvRank, MPI_COMM_CART, &postStatus[1]);

            out << "ZONE T = \"" << rank << "\", I=" << chunck[COORDINATE::X] << ", J=" << chunck[COORDINATE::Y] << ", K=" << chunck[COORDINATE::Z] << ", F=POINT" << std::endl;
            int counter = 0;
            for (unsigned k = 0; k < chunck[COORDINATE::Z]; ++k)
                for (unsigned j = 0; j < chunck[COORDINATE::Y]; ++j)
                    for (unsigned i = 0; i < chunck[COORDINATE::X]; ++i)
                    {
                        out << std::scientific << std::setprecision(5) << std::setw(15) << (coordinates3DFromReceivedRank[COORDINATE::X] * (chunck[COORDINATE::X] - 1) + i) * spacing[COORDINATE::X];
                        out << std::scientific << std::setprecision(5) << std::setw(15) << (coordinates3DFromReceivedRank[COORDINATE::Y] * (chunck[COORDINATE::Y] - 1) + j) * spacing[COORDINATE::Y];
                        out << std::scientific << std::setprecision(5) << std::setw(15) << (coordinates3DFromReceivedRank[COORDINATE::Z] * (chunck[COORDINATE::Z] - 1) + k) * spacing[COORDINATE::Z];
                        out << std::scientific << std::setprecision(5) << std::setw(15) << receiveBufferPostProcess[counter++];
                        out << std::fixed << std::setw(5) << recvRank << std::endl;
                    }
        }
        out.close();
    }
    if (size == 1)
    {
        std::ofstream out("output/out.dat");
        out << "TITLE=\"out\"" << std::endl;
        out << "VARIABLES = \"X\", \"Y\", \"Z\", \"T\"" << std::endl;
        out << "ZONE T = \"" << rank << "\", I=" << chunck[COORDINATE::X] << ", J=" << chunck[COORDINATE::Y] << ", K=" << chunck[COORDINATE::Z] << ", F=POINT" << std::endl;
        for (unsigned k = 0; k < chunck[COORDINATE::Z]; ++k)
            for (unsigned j = 0; j < chunck[COORDINATE::Y]; ++j)
                for (unsigned i = 0; i < chunck[COORDINATE::X]; ++i)
                {
                    out << std::scientific << std::setprecision(5) << std::setw(15) << (coordinates3D[COORDINATE::X] * (chunck[COORDINATE::X] - 1) + i) * spacing[COORDINATE::X];
                    out << std::scientific << std::setprecision(5) << std::setw(15) << (coordinates3D[COORDINATE::Y] * (chunck[COORDINATE::Y] - 1) + j) * spacing[COORDINATE::Y];
                    out << std::scientific << std::setprecision(5) << std::setw(15) << (coordinates3D[COORDINATE::Z] * (chunck[COORDINATE::Z] - 1) + k) * spacing[COORDINATE::Z];
                    out << std::scientific << std::setprecision(5) << std::setw(15) << T[i][j][k] << std::endl;
                }
        out.close();
    }



    MPI_Finalize();

    return 0;
}